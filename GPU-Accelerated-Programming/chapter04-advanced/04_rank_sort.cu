
#include "hip/hip_runtime.h"

#include <stdio.h>

#define arraySize 5
#define threadPerBlock 5
__global__ void addKernel(int *d_a, int *d_b)
{
	int count = 0;
	int tid = threadIdx.x;
	int ttid = blockIdx.x * threadPerBlock + tid;
	int val = d_a[ttid];
	__shared__ int cache[threadPerBlock];
	for (int i = tid; i < arraySize; i += threadPerBlock) {
		cache[tid] = d_a[i];
		__syncthreads();
		for (int j = 0; j < threadPerBlock; ++j)
			if (val > cache[j])
				count++;
		__syncthreads();
	}
	d_b[count] = val;
}

int main()
{
    
    int h_a[arraySize] = { 5, 9, 3, 4, 8 };
    int h_b[arraySize];
	int *d_a, *d_b;
	 
     
    hipMalloc((void**)&d_b, arraySize * sizeof(int));
    hipMalloc((void**)&d_a, arraySize * sizeof(int));
    
    

    // Copy input vector from host memory to GPU buffers.
    hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<arraySize/threadPerBlock, threadPerBlock>>>(d_a, d_b);

    
    hipDeviceSynchronize();
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(h_b, d_b, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	printf("The Enumeration sorted Array is: \n");
	for (int i = 0; i < arraySize; i++) {
		printf("%d\n", h_b[i]);
	}
    
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}
